
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

using namespace std;

__global__ void kernel_sum (int *A, int *B, int *C, int n); 

void sum (int *A, int *B, int *C, int n);

int main()
{
	int n;
	cout<<"Enter n:";
	cin>>n;

	int size=n*sizeof(int);
	int *deviceA,*deviceB,*deviceC;
	int *hostA = (int*)malloc(size);
	int *hostB = (int*)malloc(size);
	int *hostC = (int*)malloc(size);
	hipEvent_t start,end,start1,end1;
	hipEventCreate(&start);
	hipEventCreate(&end);
	hipEventCreate(&start1);
	hipEventCreate(&end1);


	for(int i=0;i<n;i++)
	{
		hostA[i]=rand()%n;
		hostB[i]=rand()%n;
	}

	cout<<"\nHost A:\n";
	for(int i=0;i<n;i++)
	{
		cout<<hostA[i]<<"\t";
	}

	cout<<"\nHost B:\n";
	for(int i=0;i<n;i++)
	{
		cout<<hostB[i]<<"\t";
	}

	float t=0,t1=0;
	hipEventRecord(start);
	cout<<"\nSequential processing result:\n";
	for(int i=0;i<n;i++)
	{
		cout<<hostA[i]+hostB[i]<<"\t";
	}

	hipEventRecord(end);
	hipEventSynchronize(end);
	hipEventElapsedTime(&t,start,end);

	hipMalloc(&deviceA,size);
	hipMalloc(&deviceB,size);
	hipMalloc(&deviceC,size);


	hipMemcpy(deviceA,hostA,size,hipMemcpyHostToDevice);
	hipMemcpy(deviceB,hostB,size,hipMemcpyHostToDevice);

	hipEventRecord(start1);

	sum(deviceA,deviceB,deviceC,n);

	hipEventRecord(end1);
	hipEventSynchronize(end1);
	hipEventElapsedTime(&t1,start1,end1);

	hipMemcpy(hostC,deviceC,size,hipMemcpyDeviceToHost);

	cout<<"\nParallel Execution:\nExpected\tActual\n\n";
	for(int i=0;i<n;i++)
	{
		cout<<hostA[i]+hostB[i]<<"\t\t"<<hostC[i]<<"\n";
	}
	cout<<"\n";

	cout<<"\nSequential time:"<<t;
	cout<<"\nParallel time:"<<t1;

	hipFree(deviceA);
	hipFree(deviceB);
	hipFree(deviceC);

	return hipDeviceSynchronize();
}

void sum(int *A, int *B, int *C, int n)
{
	int threadsPerBlock, blocksPerGrid;
	if(n<512)
	{
		threadsPerBlock = n;
		blocksPerGrid = 1;
	}
	else
	{
		threadsPerBlock = 512;
		blocksPerGrid = ceil(double(n)/double(threadsPerBlock));
	}
	kernel_sum<<<blocksPerGrid,threadsPerBlock>>>(A,B,C,n);
}

__global__ void kernel_sum (int *A, int *B, int *C, int n)
{
	int index=blockDim.x * blockIdx.x + threadIdx.x;

	if(index<n)
		C[index] = A[index] + B[index];
} 


