
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <cstdlib>

using namespace std;	


__global__ void kernel_multiply(int *A, int *B, int *C, int n)
{
	int row = blockDim.y * blockIdx.y + threadIdx.y;
	int col = blockDim.x * blockIdx.x + threadIdx.x;
	int sum = 0;
	if(row<n && col<n)
	{
		for(int i=0;i<n;i++)
		{
			sum+=A[row*n+i]*B[i*n+col]; 
		}
		__syncthreads();
		C[row*n+col] = sum;
	}
}

void mm(int *A, int *B, int *C, int n)
{
	dim3 threadsPerBlock(n,n);
	dim3 blocksPerGrid(1,1);
	if(n*n>512)
	{
		threadsPerBlock.x=512;
		threadsPerBlock.y=512;
		blocksPerGrid.x=ceil(double(n)/double(threadsPerBlock.x));
		blocksPerGrid.y=ceil(double(n)/double(threadsPerBlock.y));
	}
	kernel_multiply<<<blocksPerGrid,threadsPerBlock>>>(A,B,C,n);
}

int main()
{
	int n;
	cout<<"\nEnter n:";
	cin>>n;
	int size=n*sizeof(int);

	int *hostA = (int*)malloc(size*n);
	int *hostB = (int*)malloc(size*n);
	int *hostC = (int*)malloc(size*n);
	int *ans = (int*)malloc(size*n);

	for(int i=0;i<n;i++)
	{
		for(int j=0;j<n;j++)
		{
			hostA[i*n+j] = rand()%n;
		}
	}

	for(int i=0;i<n;i++)
	{
		for(int j=0;j<n;j++)
		{
			hostB[i*n+j] = rand()%n;
		}
	}

	cout<<"\nMatrix A:\n";
	for(int i=0;i<n;i++)
	{
		for(int j=0;j<n;j++)
		{
			cout<<hostA[i*n+j]<<"\t";
		}
		cout<<endl;
	}
	cout<<endl;

	cout<<"\nMatrix B:\n";
	for(int i=0;i<n;i++)
	{
		for(int j=0;j<n;j++)
		{
			cout<<hostB[i*n+j]<<"\t";
		}
		cout<<endl;
	}
	cout<<endl;

	int sum=0;
	for(int row=0;row<n;row++)
	{
		for(int col=0;col<n;col++)
		{
			sum=0;
			for(int i=0;i<n;i++)
			{
				sum+=hostA[row*n+i]*hostB[i*n+col];
			}
			ans[row*n+col] = sum;
		}
	}


	int *deviceA,*deviceB,*deviceC;

	hipMalloc(&deviceA,size*n);
	hipMalloc(&deviceB,size*n);
	hipMalloc(&deviceC,size*n);

	hipMemcpy(deviceA,hostA,size*n,hipMemcpyHostToDevice);
	hipMemcpy(deviceB,hostB,size*n,hipMemcpyHostToDevice);

	mm(deviceA,deviceB,deviceC,n);

	hipMemcpy(hostC,deviceC,size*n,hipMemcpyDeviceToHost);

	cout<<"\nAnswer=\n";
	for(int i=0;i<n*n;i++)
	{
		cout<<"( "<<i<<" )\tE = "<<ans[i]<<"\tA = "<<hostC[i]<<endl;
	}

	hipFree(deviceA);
	hipFree(deviceB);
	hipFree(deviceC);

	return hipDeviceSynchronize();
}
