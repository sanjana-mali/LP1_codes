
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>
#include <math.h>

using namespace std;
__global__ void maximum(int *a, int *b, int n)
{
	int max=0;
	int index = 256 * blockIdx.x;
	for(int i=index;i<min(256+index,n);i++)
	{
		if(a[i]>max)
		{
			max=a[i];
		}
	}
	b[blockIdx.x]=max;
}

__global__ void minimum(int *a, int *b, int n)
{
	int mini=999999;
	int index = 256 * blockIdx.x;
	for(int i=index;i<min(256+index,n);i++)
	{
		if(a[i]<mini)
		{
			mini=a[i];
		}
	}
	b[blockIdx.x]=mini;
}

__global__ void sum(int *a, int *b, int n)
{
	int sum=0;
	int index = 256 * blockIdx.x;
	for(int i=index;i<min(256+index,n);i++)
	{
		sum+=a[i];
	}
	b[blockIdx.x]=sum;
}

__global__ void standard_deviation(int *a, int *b, int n,double mean)
{
	double sum=0;
	int index = 256 * blockIdx.x;
	for(int i=index;i<min(256+index,n);i++)
	{
		sum+=(a[i]-mean)*(a[i]-mean);
	}
	b[blockIdx.x]=sum;
}

int main()
{
	int n,nmin,nsum,nmean,nsd;
	cout<<"\nEnter n:";
	cin>>n;
	int *hostA=(int*)malloc(sizeof(int)*n);

	nmin=n;
	nsum=n;
	nmean=n;
	nsd=n;

	float timeS=0.0,timeP=0.0;

	hipEvent_t startS,endS,startP,endP;
	hipEventCreate(&startS);
	hipEventCreate(&endS);
	hipEventCreate(&startP);
	hipEventCreate(&endP);

	hipEventRecord(startS);

	int max=0,min=9999999,ssum=0;
	double sd_sum=0.0f;
	for(int i=0;i<n;i++)
	{
		hostA[i]=rand()%n;
		cout<<hostA[i]<<"\t";
		if(hostA[i]>max)
		{
			max=hostA[i];
		}

		if(hostA[i]<min)
		{
			min=hostA[i];
		}
		ssum+=hostA[i];
	}
	double means=(ssum*1.0f)/(nmean*1.0f);
	for(int i=0;i<n;i++)
	{
		sd_sum+=(hostA[i]-means)*(hostA[i]-means);
	}

	hipEventRecord(endS);
	hipEventSynchronize(endS);
	hipEventElapsedTime(&timeS,startS,endS);


	hipEventRecord(startP);

	int *deviceA;
	hipMalloc(&deviceA, sizeof(int)*n);
	hipMemcpy(deviceA,hostA,sizeof(int)*n,hipMemcpyHostToDevice);

	int *deviceB;	
	int grids=ceil(n*1.0f/256*1.0f);
	hipMalloc(&deviceB, sizeof(int)*grids);

	dim3 grid(grids,1);
	dim3 block(1,1);

	while(n>1)
	{
		maximum<<<grid,block>>>(deviceA,deviceB,n);
		n=ceil(n*1.0f/256*1.0f);
		hipMemcpy(deviceA,deviceB,n*sizeof(int),hipMemcpyDeviceToDevice);
	}

	int maxi[2];
	hipMemcpy(maxi,deviceA,4,hipMemcpyDeviceToHost);
	cout<<"\nParallel Max="<<maxi[0]<<endl<<"Sequential Max="<<max<<endl;

//----------------------------------min----------------------------------------------------------

	hipMemcpy(deviceA,hostA,sizeof(int)*nmin,hipMemcpyHostToDevice);
	while(nmin>1)
	{
		minimum<<<grid,block>>>(deviceA,deviceB,nmin);
		nmin=ceil(nmin*1.0f/256*1.0f);
		hipMemcpy(deviceA,deviceB,nmin*sizeof(int),hipMemcpyDeviceToDevice);
	}

	int mini[2];
	hipMemcpy(mini,deviceA,4,hipMemcpyDeviceToHost);
	cout<<"\nParallel Min="<<mini[0]<<endl<<"Sequential Min="<<min<<endl;

//--------------------------------sum------------------------------------------------------------

	hipMemcpy(deviceA,hostA,sizeof(int)*nsum,hipMemcpyHostToDevice);
	while(nsum>1)
	{
		sum<<<grid,block>>>(deviceA,deviceB,nsum);
		nsum=ceil(nsum*1.0f/256*1.0f);
		hipMemcpy(deviceA,deviceB,nsum*sizeof(int),hipMemcpyDeviceToDevice);
	}

	int sums[2];
	hipMemcpy(sums,deviceA,4,hipMemcpyDeviceToHost);
	cout<<"\nParallel Sum="<<sums[0]<<endl<<"Sequential sum="<<ssum<<endl;

	double mean=(double)(sums[0]*1.0f/nmean*1.0f);

	cout<<"\nParallel Mean="<<mean<<endl<<"Sequential mean="<<means<<endl;


//--------------------------------sd-------------------------------------------------------------

	hipMemcpy(deviceA,hostA,sizeof(int)*nsd,hipMemcpyHostToDevice);
	while(nsd>1)
	{
		standard_deviation<<<grid,block>>>(deviceA,deviceB,nsd,mean);
		nsd=ceil(nsd*1.0f/256*1.0f);
		hipMemcpy(deviceA,deviceB,nsd*sizeof(int),hipMemcpyDeviceToDevice);
	}

	int sdp[2];
	hipMemcpy(sdp,deviceA,4,hipMemcpyDeviceToHost);
	cout<<"\nParallel SD="<<(double)sqrt((sdp[0]*1.0f)/(nmean*1.0f))<<endl<<"Sequential SD="<<(double)sqrt((sd_sum*1.0f)/(nmean*1.0f))<<endl;


	hipEventRecord(endP);
	hipEventSynchronize(endP);
	hipEventElapsedTime(&timeP,startP,endP);

	cout<<"\nSequential Time="<<timeS;
	cout<<"\nParallel Time="<<timeP<<endl;

	return hipDeviceSynchronize();
}
