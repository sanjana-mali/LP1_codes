
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

using namespace std;

__global__ void min1(int *a,int *b,int n)
{
	int index=256*blockIdx.x;
	int mini=999999;
	for(int i=index;i<min(256+index,n);i++)
	{
		if(a[i]<mini)
		{
			mini=a[i];
		}
	}
	b[blockIdx.x]=mini;
}

int main()
{
	int n=0;
	cout<<"Enter n:";
	cin>>n;
	int *a=(int*)malloc(n*sizeof(int));
	int max=0;

	for(int i=0;i<n;i++)
	{
		a[i]=rand()%n;
		cout<<a[i]<<"\t";
	}

	for(int i=0;i<n;i++)
	{
		if(a[i]>max)
		{
			max=a[i];
		}
	}
	//cout<<"\nMax="<<max<<endl;

	int min=999999;
	for(int i=0;i<n;i++)
	{
		if(a[i]<min)
		{
			min=a[i];
		}
	}
	cout<<"\nMin="<<min<<endl;

	int *deviceA,*deviceB;

	int grids=ceil(n*1.0f/256*1.0f);
	hipMalloc(&deviceA,n*sizeof(int));
	hipMemcpy(deviceA,a,n*sizeof(int),hipMemcpyHostToDevice);

	dim3 grid(grids,1);
	dim3 block(1,1);

	hipMalloc(&deviceB,grids*sizeof(int));

	while(n>1)
	{
		min1<<<grid,block>>>(deviceA,deviceB,n);
		n=ceil(n*1.0f/256*1.0f);
		hipMemcpy(deviceA,deviceB,n*sizeof(int),hipMemcpyDeviceToDevice);
	}

	int ans[2];
	hipMemcpy(ans,deviceA,4,hipMemcpyDeviceToHost);

	cout<<"\nParallel Min="<<ans[0]<<endl;
	return hipDeviceSynchronize();
}
