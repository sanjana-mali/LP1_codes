
#include <hip/hip_runtime.h>
#include<iostream>
#include<cstdio>

using namespace std;


__global__ void var(int *a,int *b,int n,float mean)
{
	int block=256*blockIdx.x;
	double sum=0.0f;
	for(int i=block;i<min(block+256,n);i++)
	{
		sum=sum+((a[i]-mean)*(a[i]-mean));
	}
	b[blockIdx.x]=sum;
}

__global__ void sum(int *a,int *b,int n)
{
	int block=256*blockIdx.x;
	int sum=0;
	for(int i=block;i<min(block+256,n);i++)
	{
		sum=sum+a[i];
	}
	b[blockIdx.x]=sum;
}

int main()
{
	cout<<"Enter the no of elements:";
	int n;
	cin>>n;
	int n1=n,p1=n;
	int *a=(int*)malloc(n*sizeof(int));
	for(int i=0;i<n;i++)
	{
		a[i]=rand()%100;
		cout<<a[i]<<"\t";
	}

	hipEvent_t start1,end1;
	hipEventCreate(&start1);
	hipEventCreate(&end1);
	hipEventRecord(start1);
	int sum1=0;
	for(int i=0;i<n;i++)
	{
		sum1+=a[i];
	}
	float mean1=0.0f;
	mean1=sum1/(n*1.0f);
	double s=0.0f;
	for(int i=0;i<n;i++)
	{
		s=s+((a[i]-mean1)*(a[i]-mean1));
	}
	double sd1=sqrt(s/n*1.0f);
	cout<<"\nAdd="<<s;
	hipEventRecord(end1);
	hipEventSynchronize(end1);
	float time1=0;
	hipEventElapsedTime(&time1,start1,end1);
	cout<<"\nSequential Processing:";
	cout<<"\nSum="<<sum1;
	cout<<"\nMean="<<mean1;
	cout<<"\nStandard deviation="<<sd1;
	cout<<"\nSequential time="<<time1<<endl;

	int *ad,*bd;
	int size=n*sizeof(int);
	hipMalloc(&ad,size);
	hipMemcpy(ad,a,size,hipMemcpyHostToDevice);
	int grids=ceil(n*1.0f/256.0f);
	hipMalloc(&bd,grids*sizeof(int));
	dim3 grid(grids,1);
	dim3 block(1,1);
	int p=n;
	hipEvent_t start,end;
	hipEventCreate(&start);
	hipEventCreate(&end);
	hipEventRecord(start);
	while(n>1)
	{

		sum<<<grid,block>>>(ad,bd,n);
		n=ceil(n*1.0f/256.0f);
		hipMemcpy(ad,bd,n*sizeof(int),hipMemcpyDeviceToDevice);

	}
	hipEventRecord(end);
	hipEventSynchronize(end);
	float time=0;
	hipEventElapsedTime(&time,start,end);
	int add[2];
	n=p;
	hipMemcpy(add,ad,4,hipMemcpyDeviceToHost);
	cout<<"\nSum="<<add[0]<<endl;
	float mean=0.0f;
	mean=add[0]/(n*1.0f);
	cout<<"Mean="<<mean<<endl;


	int *ad1,*bd1;

	hipMalloc(&ad1,size);
	hipMemcpy(ad1,a,size,hipMemcpyHostToDevice);
	int grids1=ceil(n1*1.0f/256.0f);
	hipMalloc(&bd1,grids1*sizeof(int));
	dim3 grid1(grids1,1);
	dim3 block1(1,1);

	//var<<<grid,block>>>(ad,bd,n,mean);
	//n=ceil(n*1.0f/256.0f);	
	//sum<<<grid,block>>>(bd,ad,n);

	while(n1>1)
	{

		var<<<grid1,block1>>>(ad1,bd1,n1,mean);
		n1=ceil(n1*1.0f/256.0f);
		hipMemcpy(ad1,bd1,n1*sizeof(int),hipMemcpyDeviceToDevice);

	}

	long long int add1[2];
	hipMemcpy(add1,ad1,4,hipMemcpyDeviceToHost);
	cout<<"\nAdd="<<add1[0]<<endl;
	float sd_=sqrt(add1[0]/(p1*1.0f));
	cout<<"Standard deviation="<<sd_<<endl;
	cout<<"Parallel time="<<time<<endl;
	return hipDeviceSynchronize();
}
